#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"           //CUDA运行时API  
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include ""     
#include <malloc.h>
#include <stdio.h> 
#include <stdlib.h>
#include <sys/time.h>

#define CAM_W 1920
#define CAM_H 1080
#define BLOCK_NUM  64
#define THREAD_NUM 512

#if 1
static struct timeval tv0;
static struct timeval tv1;
static struct timezone tz;
#endif
__global__ void cudaResizeLinear(float *src, float *dst, int w0, int h0, int w1, int h1)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;

	int y1,y2, x1,x2,  x, y;
	float fx1, fx2, fy1, fy2;

	for (i = bid * THREAD_NUM + tid; i < w1 * h1; i += BLOCK_NUM * THREAD_NUM)
	{
		x = (i) % w1;
		y = (i) / w1;

		x1 = (int)(x* ((float)w0 / (float)w1));
		x2 = (int)(x* ((float)w0 / (float)w1)) + 1;
		y1 = (int)(y* ((float)h0 / (float)h1));
		y2 = (int)(y* ((float)h0 / (float)h1)) + 1;

		fx1 = (((float)x* (((float)w0) / (float)w1))) - (int)(x * (((float)w0) / (float)w1));
		fx2 = 1.0f - fx1;
		fy1 = (((float)y* (((float)h0) / (float)h1))) - (int)(y * (((float)h0) / (float)h1));
		fy2 = 1.0f - fy1;

		float s1 = fx1*fy1;
		float s2 = fx2*fy1;
		float s3 = fx2*fy2;
		float s4 = fx1*fy2;

		dst[i * 3 + 0] = (src[y1 * w0 * 3 + x1 * 3 + 0]) * s3 + (src[y1 * w0 *3 + x2*3 + 0]) * s4 + (src[y2 * w0*3 + x1*3 + 0]) * s2 + (src[y2 * w0 *3 + x2 *3 + 0]) * s1;
		dst[i * 3 + 1] = (src[y1 * w0 * 3 + x1 * 3 + 1]) * s3 + (src[y1 * w0 *3 + x2*3 + 1]) * s4 + (src[y2 * w0*3 + x1*3 + 1]) * s2 + (src[y2 * w0 *3 + x2 *3 + 1]) * s1;
		dst[i * 3 + 2] = (src[y1 * w0 * 3 + x1 * 3 + 2]) * s3 + (src[y1 * w0 *3 + x2*3 + 2]) * s4 + (src[y2 * w0*3 + x1*3 + 2]) * s2 + (src[y2 * w0 *3 + x2 *3 + 2]) * s1;
	}
}

__global__ void addKernel(float *a,  const char *b, int w, int h)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i;

    for (i = bid * THREAD_NUM + tid; i < w * h / 2; i += BLOCK_NUM * THREAD_NUM) {

        a[i*6 + 0]=(10000*b[i*4 + 1]+14075*(b[i*4 + 2]-128))/10000;
        a[i*6 + 1]=(10000*b[i*4 + 1]-3455*( b[i*4 + 0]-128)-7169*(b[i*4 + 2]-128))/10000;
        a[i*6 + 2]=(10000*b[i*4 + 1]+17990*(b[i*4 + 0]-128))/10000;
        a[i*6 + 3]=(10000*b[i*4 + 3]+14075*(b[i*4 + 2]-128))/10000;
        a[i*6 + 4]=(10000*b[i*4 + 3]-3455*( b[i*4 + 0]-128)-7169*(b[i*4 + 2]-128))/10000;
        a[i*6 + 5]=(10000*b[i*4 + 3]+17990*(b[i*4 + 0]-128))/10000;

        if(a[i*6 + 0]>255) a[i*6 + 0]=255; if(a[i*6 + 0]<0) a[i*6 + 0]=0;
        if(a[i*6 + 1]>255) a[i*6 + 1]=255; if(a[i*6 + 1]<0) a[i*6 + 1]=0;
        if(a[i*6 + 2]>255) a[i*6 + 2]=255; if(a[i*6 + 2]<0) a[i*6 + 2]=0;
        if(a[i*6 + 3]>255) a[i*6 + 3]=255; if(a[i*6 + 3]<0) a[i*6 + 3]=0;
        if(a[i*6 + 4]>255) a[i*6 + 4]=255; if(a[i*6 + 4]<0) a[i*6 + 4]=0;
        if(a[i*6 + 5]>255) a[i*6 + 5]=255; if(a[i*6 + 5]<0) a[i*6 + 5]=0;

    }
}

__global__ void cudasplice(char *src0, char *src1, char *src2, char * src3, char *dst, int w, int h)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i, h1, w1;
    int size = w * h;
    int bw = 2 * w;

    for (i = bid * THREAD_NUM + tid; i < size * 4; i += BLOCK_NUM * THREAD_NUM) {
        w1 = i % bw;
        h1 = i / bw;
        if (((i % bw) < w ) && (i / bw < h)) {
            dst[i * 2 + 0] = src0[(h1 * w + w1) * 2 + 0];
            dst[i * 2 + 1] = src0[(h1 * w + w1) * 2 + 1];
        } else if ((i % bw >= w ) && (i / bw < h)) {
            dst[i * 2 + 0] = src1[(h1 * w + w1) * 2 + 0];
            dst[i * 2 + 1] = src1[(h1 * w + w1) * 2 + 1];
        } else if ((i % bw < w ) && (i / bw >= h)) {
            h1 = h1 - h;
            dst[i * 2 + 0] = src2[(h1 * w + w1) * 2 + 0];
            dst[i * 2 + 1] = src2[(h1 * w + w1) * 2 + 1];
        } else if ((i % bw >= w ) && (i / bw >= h)) {
            h1 = h1 - h;
            dst[i * 2 + 0] = src3[(h1 * w + w1) * 2 + 0];
            dst[i * 2 + 1] = src3[(h1 * w + w1) * 2 + 1];
        } else {
            printf(" Should not be here\n");
        }
    }
}
__global__ void cudaShowconvert(char *dst, const float *src, int w, int h)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i;

    for (i = bid * THREAD_NUM + tid; i < w * h; i += BLOCK_NUM * THREAD_NUM) {
        dst[i * 3 + 0] = src[i * 3 + 2];
        dst[i * 3 + 1] = src[i * 3 + 1];
        dst[i * 3 + 2] = src[i * 3 + 0];
    }
}

__global__ void cudabgr2rgb(float *dst, const float *src, int w, int h)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i, j, c;

	for (c = 0; c < 3; c++)
	for (j = 0; j < h; j++)
    for (i = bid * THREAD_NUM + tid; i < w; i += BLOCK_NUM * THREAD_NUM) {
        dst[i  + j *  w  + c * h * w] = src[i * 3 + 3 * w * j + c] / 255.;
    }
}


__global__ void cudaswapfloat(float *dst, int w, int h)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i;
	float swap;

    for (i = bid * THREAD_NUM + tid; i < w * h; i += BLOCK_NUM * THREAD_NUM) {
		swap = dst[i];
		dst[i] = dst[i + w*h*2];
		dst[i + w*h*2] = swap;
    }
}

extern "C"
bool InitCUDA(void)
{
    int count = 0;
    int i = 0;
    hipGetDeviceCount(&count); //看看有多少个设备?
    if(count == 0)   //哈哈~~没有设备.
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
    hipDeviceProp_t prop;
    for(i = 0; i < count; i++)  //逐个列出设备属性:
    {
        if(hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if(prop.major >= 1)
            {
                break;
            }
        }
    }
    if(i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA.\n");
        return false;
    }
    hipSetDevice(i);

    hipDeviceProp_t sDevProp = prop;

    printf( "\n\nGPU Num: %d \n", i);
    printf( "Device name: %s\n", sDevProp.name );
    printf( "Device memory: %lu\n", sDevProp.totalGlobalMem );
    printf( "Memory per-block: %lu\n", sDevProp.sharedMemPerBlock );
    printf( "Register per-block: %u\n", sDevProp.regsPerBlock );
    printf( "Warp size: %u\n", sDevProp.warpSize );
    printf( "Memory pitch: %lu\n", sDevProp.memPitch );
    printf( "Constant Memory: %lu\n", sDevProp.totalConstMem );
    printf( "Max thread per-block: %u\n", sDevProp.maxThreadsPerBlock );
    printf( "Max thread dim: ( %d, %d, %d )\n", sDevProp.maxThreadsDim[0],
            sDevProp.maxThreadsDim[1], sDevProp.maxThreadsDim[2] );
    printf( "Max grid size: ( %d, %d, %d )\n", sDevProp.maxGridSize[0],  
            sDevProp.maxGridSize[1], sDevProp.maxGridSize[2] );
    printf( "Ver: %d.%d\n", sDevProp.major, sDevProp.minor );
    printf( "Clock: %d\n", sDevProp.clockRate );
    printf( "textureAlignment: %lu\n", sDevProp.textureAlignment );
    printf( "CUDART_VERSION: %d\n", CUDART_VERSION);

    if (!prop.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", i);

    } else {
        printf("Device %d support mapping CPU host memory!\n", i);
    }

    hipSetDeviceFlags(hipDeviceMapHost);


    printf("\nCUDA initialized.\n\n");
    return true;
}

extern "C"
void cudayuv2rgb(float *dev_a, const char *dev_b, int w, int h)
{
    gettimeofday(&tv0, &tz);
	addKernel<<<BLOCK_NUM, THREAD_NUM>>>(dev_a, dev_b, w, h);
    gettimeofday(&tv1, &tz);
    //printf("\n kernel running Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}

extern "C"
void cuda_resize(float *src, float *dst, int src_w, int src_h, int dst_w, int dst_h)
{
    gettimeofday(&tv0, &tz);
	cudaResizeLinear<<<BLOCK_NUM, THREAD_NUM>>>(src, dst, src_w, src_h, dst_w, dst_h);
    gettimeofday(&tv1, &tz);
    //printf("\n kernel running Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}

extern "C"
void cudaShowCon(char *dst, const float *src, int w, int h)
{
    gettimeofday(&tv0, &tz);
    cudaShowconvert<<<BLOCK_NUM, THREAD_NUM>>>(dst, src, w, h);
    gettimeofday(&tv1, &tz);
//    printf("\n kernel show convert Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}
extern "C"
void cudabgrtorgb(float *dst, const float *src, int w, int h)
{
    gettimeofday(&tv0, &tz);
    cudabgr2rgb<<<BLOCK_NUM, THREAD_NUM>>>(dst, src, w, h);
    gettimeofday(&tv1, &tz);
//    printf("\n kernel show convert Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}
extern "C"
void cudaswap(float *dst, int w, int h)
{
    gettimeofday(&tv0, &tz);
    cudaswapfloat<<<BLOCK_NUM, THREAD_NUM>>>(dst,w, h);
    gettimeofday(&tv1, &tz);
//    printf("\n kernel show convert Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}

extern "C"
void cuda_splice(char *src, int w, int h)
{
    int offset = w * h * 2;
    gettimeofday(&tv0, &tz);
    cudasplice<<<BLOCK_NUM, THREAD_NUM>>>(src, src + offset * 1, src + offset * 2, src + offset * 3, src + offset * 4, w, h);
    gettimeofday(&tv1, &tz);
//    printf("\n kernel show convert Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}

extern "C"
void cuda_splice_four(char *src0, char *src1, char *src2, char *src3, char *dst, int w, int h)
{
    //int offset = w * h * 2;
    gettimeofday(&tv0, &tz);
    cudasplice<<<BLOCK_NUM, THREAD_NUM>>>(src0, src1, src2, src3, dst, w, h);
    gettimeofday(&tv1, &tz);
//    printf("\n kernel show convert Cost time :  %lu us\n", tv1.tv_usec - tv0.tv_usec);
}
